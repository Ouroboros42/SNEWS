#include "hip/hip_runtime.h"
#include <iostream>
#include "main.cuh"
#include <chrono>

__global__ void vectorAdd(int *a, int *b, int *c) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int computeCPU(int *a, int *b, int *c, int N) {
    auto start = std::chrono::high_resolution_clock::now();
    for(int i = 0; i < N; i++) {
        c[i] = a[i] + b[i];
    }
    auto stop = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    return elapsed.count();
}

int computeGPU(int *a, int *b, int *c, int N) {

    int *dev_a, *dev_b, *dev_c;

    size_t sizeOfArrayInBytes = N * sizeof(int);

    hipMalloc((void** )&dev_a, sizeOfArrayInBytes);
    hipMalloc((void** )&dev_b, sizeOfArrayInBytes);
    hipMalloc((void** )&dev_c, sizeOfArrayInBytes);

    hipMemcpy(dev_a, a, sizeOfArrayInBytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeOfArrayInBytes, hipMemcpyHostToDevice);

    // Time GPU Kernal
    auto start = std::chrono::high_resolution_clock::now();

    const int NUM_BLOCKS = 1;
    const int NUM_THREADS = N;
    vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(dev_a, dev_b, dev_c);
    hipDeviceSynchronize();

    auto stop = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

    hipMemcpy(c, dev_c, sizeOfArrayInBytes, hipMemcpyDeviceToHost);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return elapsed.count();
}


int main() {
    const int N = 100;
    int a[N], b[N], c_cpu[N], c_gpu[N];

    for(int i = 0; i < N; i++) {
        a[i] = 1;
        b[i] = 1;
    }

    int cpuTime = computeCPU(a, b, c_cpu, N);
    int gpuTime = computeGPU(a, b, c_gpu, N);

    std::cout << "CPU: " << cpuTime << std::endl;
    std::cout << "GPU: " << gpuTime << std::endl;
}



