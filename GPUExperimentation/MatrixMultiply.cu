
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

// CUDA kernel for matrix multiplication
__global__ void matrixMultiply(int *a, int *b, int *c, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int sum = 0;
        for (int k = 0; k < N; ++k) {
            sum += a[row * N + k] * b[k * N + col];
        }
        c[row * N + col] = sum;
    }
}

int main() {
    const int N = 4;
    int a[N][N], b[N][N], c[N][N];

    int *d_a, *d_b, *d_c;

    int size = N * N * sizeof(int); // Size of matrix elements in bytes

    // Initialize matrices a and b
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            a[i][j] = i + j;
            b[i][j] = i - j;
        }
    }

    // Allocate memory for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + 15) / 16, (N + 15) / 16);

    // Launch kernel to perform matrix multiplication
    matrixMultiply<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Output result matrix
    std::cout << "Result Matrix:" << std::endl;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << c[i][j] << "\t";
        }
        std::cout << std::endl;
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
