
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <chrono>


__global__ void matrixMultiply(int *a, int *b, int *c, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k) {
            sum += a[row * N + k] * b[k * N + col];
        }
        c[row*N + col] = sum;
    }
}


int computeCPU(int *a, int *b, int *c, int N) {
    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            int sum = 0;
            for (int k = 0; k < N; k++) {
                sum += a[i * N + k] * b[k * N + j];
            }
            c[i * N + j] = sum;
        }
    }
    auto stop = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    return elapsed.count();
}



int computeGPU(int *a, int *b, int *c, int N) {

    // allocate memory for gpu
    int *dev_a, *dev_b, *dev_c;
    size_t sizeOfMatrixInBytes = N * N * sizeof(int);

    hipMalloc((void **)&dev_a, sizeOfMatrixInBytes);
    hipMalloc((void **)&dev_b, sizeOfMatrixInBytes);
    hipMalloc((void **)&dev_c, sizeOfMatrixInBytes);

    hipError_t e1 =  hipMemcpy(dev_a, a, sizeOfMatrixInBytes, hipMemcpyHostToDevice);
    hipError_t e2 = hipMemcpy(dev_b, b, sizeOfMatrixInBytes, hipMemcpyHostToDevice);

    if (e1 != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(e1) << std::endl;
    }
    if (e2 != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(e2) << std::endl;
    }

    // Define grid and block dimensions
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);
    if (N*N > 512){
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = ceil(double(N)/double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(N)/double(threadsPerBlock.y));
    }

    // Time GPU Kernal
    auto start = std::chrono::high_resolution_clock::now();

    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c, N);
    hipDeviceSynchronize();

    auto stop = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

    // Copy result back to host
    hipError_t e3 = hipMemcpy(c, dev_c, sizeOfMatrixInBytes, hipMemcpyDeviceToHost);

    if (e3 != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(e3) << std::endl;
    }

    // Free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return elapsed.count();
}

bool verifyResults(int *c_cpu, int *c_gpu, int N) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (c_cpu[i * N + j] != c_gpu[i * N + j]) {
                std::cerr << "Mismatch at index " << i << ", " << j << ": " << c_cpu[i * N + j] << " != " << c_gpu[i * N + j] << std::endl;
                return false;
            }
        }
    }
    return true;
}

void printArrays(int *a, int *b, int *c_cpu, int *c_gpu, int N) {
    std::cout << "Matrix A: " << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << a[N*i + j] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << "Matrix B: " << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << b[N*i + j] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << "Matrix C (CPU): " << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << c_cpu[N*i + j] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << "Matrix C (GPU): " << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << c_gpu[N*i + j] << " ";
        }
        std::cout << std::endl;
    }
}



int main() {

    const int N = 16;
    int a[N*N], b[N*N], c_cpu[N*N], c_gpu[N*N];

    // Initialize matrices a and b
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            a[N*i + j] = i + j;
            b[N*i + j] = i - j;
        }
    }

    int cpuTime = computeCPU((int *)a, (int *)b, (int *)c_cpu, N);
    std::cout << "CPU: " << cpuTime << std::endl;

    int gpuTime = computeGPU((int *)a, (int *)b, (int *)c_gpu, N);
    std::cout << "GPU: " << gpuTime << std::endl;

    std::cout << "Verification: " << verifyResults((int *)c_cpu, (int *)c_gpu, N) << std::endl;

    //printArrays((int *)a, (int *)b, (int *)c_cpu, (int *)c_gpu, N);
}
