#include "hip/hip_runtime.h"
#include <iostream>
#include "vectorAdd.cuh"
#include <chrono>
#include <cmath>

__global__ void vectorAdd(int *a, int *b, int *c) {
    int i = threadIdx.x;
    c[i] = expf(a[i]) * expf(b[i]);
}

int computeCPU(int *a, int *b, int *c, int N) {
    auto start = std::chrono::high_resolution_clock::now();
    for(int i = 0; i < N; i++) {
        c[i] = exp(a[i]) * exp(b[i]);
    }
    auto stop = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    return elapsed.count();
}

int computeGPU(int *a, int *b, int *c, int N) {

    int *dev_a, *dev_b, *dev_c;

    size_t sizeOfArrayInBytes = N * sizeof(int);

    hipMalloc((void** )&dev_a, sizeOfArrayInBytes);
    hipMalloc((void** )&dev_b, sizeOfArrayInBytes);
    hipMalloc((void** )&dev_c, sizeOfArrayInBytes);

    // std::cout << "array a: ";
    // for(int i = 0; i < N; i++) {
    //     std::cout << a[i] << " ";
    // }
    // std::cout << std::endl;
    // std::cout << "array b: ";
    // for(int i = 0; i < N; i++) {
    //     std::cout << b[i] << " ";
    // }
    // std::cout << std::endl;

    hipError_t e1 = hipMemcpy(dev_a, a, sizeOfArrayInBytes, hipMemcpyHostToDevice);
    hipError_t e2 = hipMemcpy(dev_b, b, sizeOfArrayInBytes, hipMemcpyHostToDevice);

    // std::cout << "copy to device: " << hipGetErrorString(e1) << std::endl;
    // std::cout << "copy to device: " << hipGetErrorString(e2) << std::endl;

    // Time GPU Kernal
    auto start = std::chrono::high_resolution_clock::now();

    const int NUM_BLOCKS = 1;
    const int NUM_THREADS = N;

    vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(dev_a, dev_b, dev_c);
    hipDeviceSynchronize();

    auto stop = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

    hipError_t e3 = hipMemcpy(c, dev_c, sizeOfArrayInBytes, hipMemcpyDeviceToHost);
    // std::cout << "copy to host: " << hipGetErrorString(e3) << std::endl;

    // std::cout << "array c: ";
    // for(int i = 0; i < N; i++) {
    //     std::cout << c[i] << " ";
    // }
    // std::cout << std::endl;

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return elapsed.count();
}

int verify(int *a, int *b, int N) {
    for(int i = 0; i < N; i++) {
        if (a[i] != b[i]) {
            std::cout << "Mismatch at " << i << ": " << a[i] << " != " << b[i] << std::endl;
            return 0;
        }
    }
    return 1;
}


void singleRun() {
    const int N = 1000;
    int a[N], b[N], c_cpu[N], c_gpu[N];

    for(int i = 0; i < N; i++) {
        a[i] = i % 10;
        b[i] = i % 10;
    }

    int cpuTime = computeCPU(a, b, c_cpu, N);
    int gpuTime = computeGPU(a, b, c_gpu, N);

    std::cout << "Verification: " << verify(c_cpu, c_gpu, N) << std::endl;

    std::cout << "CPU: " << cpuTime << std::endl;
    std::cout << "GPU: " << gpuTime << std::endl;
}

int main() {
    singleRun();
    return 0;
}

